#include "hip/hip_runtime.h"
// Student: Catalin Consantin Usurelu
// Grupa: 333CA

/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.
 *
 * This software and the information contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a Non-Disclosure Agreement.  Any reproduction or
 * disclosure to any third party without the express written consent of
 * NVIDIA is prohibited.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.  This source code is a "commercial item" as
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer software" and "commercial computer software
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime_api.h>
#include <helper_timer.h>
#include <helper_functions.h>
#include <hip/hip_vector_types.h>

// includes, project
#include "2Dconvolution.h"

#define OFFSET KERNEL_SIZE / 2
#define ALLIGN_MID KERNEL_SIZE / 2


////////////////////////////////////////////////////////////////////////////////
// declarations, forward

extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int);

Matrix AllocateDeviceMatrix(int width, int height);
Matrix AllocateMatrix(int width, int height);
void FreeDeviceMatrix(Matrix* M);
void FreeMatrix(Matrix* M);

void ConvolutionOnDevice(const Matrix M, const Matrix N, Matrix P);
void ConvolutionOnDeviceShared(const Matrix M, const Matrix N, Matrix P);

// Get a matrix element
__device__ float GetElement(const Matrix A, int row, int col)
{
  return A.elements[row * A.pitch + col];
}
 
// Set a matrix element
__device__ void SetElement(Matrix A, int row, int col, float value)
{
  A.elements[row * A.pitch + col] = value;
}

////////////////////////////////////////////////////////////////////////////////
// Înmulțirea fără memorie partajată
////////////////////////////////////////////////////////////////////////////////
__global__ void ConvolutionKernel(Matrix M, Matrix N, Matrix P)
{
    // Each thread computes one element of P
    // by accumulating results into Pvalue
    
    float Pvalue = 0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Outside of range
    if(row >= N.height || col >= N.width)
    {
        return;
    }
  
    // check the start and end values of m and n to prevent overrunning the 
    // matrix edges
    unsigned int mbegin = (row < 2)? 2 - row : 0;
    unsigned int mend = (row > (N.height - 3))?
                            N.height - row + 2 : 5;
    unsigned int nbegin = (col < 2)? 2 - col : 0;
    unsigned int nend = (col > (N.width - 3))?
                            (N.width - col) + 2 : 5;
                            
    // overlay A over B centered at element (i,j).  For each 
    // overlapping element, multiply the two and accumulate
    for(unsigned int m = mbegin; m < mend; m++)
    {
        for(unsigned int n = nbegin; n < nend; n++)
        {
            Pvalue += M.elements[m * 5 + n] * 
                    N.elements[N.width * (row + m - 2) + (col + n - 2)];
        }
    }
    
    // store the result
    P.elements[row * N.width + col] = (float)Pvalue;
}


////////////////////////////////////////////////////////////////////////////////
// Înmulțirea cu memorie partajată
////////////////////////////////////////////////////////////////////////////////
__global__ void ConvolutionKernelShared(Matrix M, Matrix N, Matrix P)
{
    float Pvalue = 0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

   
    __shared__ float Ms[KERNEL_SIZE * KERNEL_SIZE];
    __shared__ float Ns[BLOCK_SIZE + KERNEL_SIZE - 1][BLOCK_SIZE + KERNEL_SIZE - 1];
   
   // Primul bloc de 5X5 threaduri copiaza matricea shared
   if(threadIdx.x < 5 && threadIdx.y < 5)
        Ms[threadIdx.y * KERNEL_SIZE + threadIdx.x] = M.elements[threadIdx.y * KERNEL_SIZE + threadIdx.x];

   __syncthreads();
   
   // ns_row, ns_col sunt defapt niste indici care repezinta blocul de threaduri
   // relativ la coltul stanga sus al lui Ns (matricea N shared)
   // matricea este centrata in mijloc (adica decalam cu 2) - practic
   // la inceput se suprapune cu blocul pentru care calculam valori
   int ns_row = threadIdx.y + ALLIGN_MID;
   int ns_col = threadIdx.x + ALLIGN_MID;
   
   // Explicatii cod in Readme
   // Aici doar folosim "blocul de threaduri" (sau parti ale lui) pentru a
   // copia parti din N in Ns
   if(row - OFFSET >= 0 && col - OFFSET >= 0)    
        Ns[ns_row - OFFSET][ns_col - OFFSET] = GetElement(N, row - OFFSET, col - OFFSET);
    
    if(threadIdx.y >= BLOCK_SIZE - 4)
    {
        if(row + OFFSET < N.height && col - OFFSET >= 0)    
            Ns[ns_row + OFFSET][ns_col - OFFSET] = GetElement(N, row + OFFSET, col - OFFSET);
            
        if(threadIdx.x >= BLOCK_SIZE - 4)
        {
            if(row + OFFSET < N.height && col + OFFSET < N.width)    
                Ns[ns_row + OFFSET][ns_col + OFFSET] = GetElement(N, row + OFFSET, col + OFFSET);
        }
    }
    
    if(threadIdx.x >= BLOCK_SIZE - 4)
         if(row - OFFSET >= 0 && col + OFFSET < N.width)    
        Ns[ns_row - OFFSET][ns_col + OFFSET] = GetElement(N, row - OFFSET, col + OFFSET);

    // Aveam nevoie de toate thread-urile pentru partea de mai sus (mai usor de implementat)
    if(row >= N.height || col >= N.width)
    {
        return;
    }
    
    // Asteptam sa se termine copierea in Ns
    __syncthreads();

    // Ne intereseaza doar "mijlocul lui Ns", adica fara margini
    // => incepem cu un offset de 2 fata de coltul stanga sus
    // Restul codului este ca la non-shared sau varianta seriala
    // doar ca accesam matricile shared
    int NsRow = threadIdx.y + KERNEL_SIZE / 2;
    int NsCol = threadIdx.x + KERNEL_SIZE / 2;
                    
    // check the start and end values of m and n to prevent overrunning the 
    // matrix edges
    unsigned int mbegin = (row < 2)? 2 - row : 0;
    unsigned int mend = (row > (N.height - 3))?
                            N.height - row + 2 : 5;
    unsigned int nbegin = (col < 2)? 2 - col : 0;
    unsigned int nend = (col > (N.width - 3))?
                            (N.width - col) + 2 : 5;
                            
    // overlay A over B centered at element (i,j).  For each 
    // overlapping element, multiply the two and accumulate
    for(unsigned int m = mbegin; m < mend; m++)
    {
        for(unsigned int n = nbegin; n < nend; n++)
        {
            Pvalue += Ms[m * 5 + n] * 
                    Ns[NsRow + m - 2][NsCol + n - 2];
        }
    }
    
    // store the result
    P.elements[row * N.width + col] = (float)Pvalue;
}

////////////////////////////////////////////////////////////////////////////////
// Returnează 1 dacă matricele sunt ~ egale
////////////////////////////////////////////////////////////////////////////////
int CompareMatrices(Matrix A, Matrix B)
{
    int i;
    if(A.width != B.width || A.height != B.height || A.pitch != B.pitch)
        return 0;
    int size = A.width * A.height;
    for(i = 0; i < size; i++)
        if(fabs(A.elements[i] - B.elements[i]) > MAX_ERR)
            return 0;
    return 1;
}
void GenerateRandomMatrix(Matrix m)
{
    int i;
    int size = m.width * m.height;

    srand(time(NULL));

    for(i = 0; i < size; i++)
        m.elements[i] = rand() / (float)RAND_MAX;
}

////////////////////////////////////////////////////////////////////////////////
// main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) 
{
    int width = 0, height = 0;
    FILE *f, *out;
    if(argc < 2)
    {
        printf("Argumente prea puține, trimiteți id-ul testului care trebuie rulat\n");
        return 0;
    }
    char name[100];
    sprintf(name, "./tests/test_%s.txt", argv[1]);
    f = fopen(name, "r");
    out = fopen("out.txt", "a");
    fscanf(f, "%d%d", &width, &height);
    Matrix M;//kernel de pe host
    Matrix N;//matrice inițială de pe host
    Matrix P;//rezultat fără memorie partajată calculat pe GPU
    Matrix PS;//rezultatul cu memorie partajată calculat pe GPU
    
    M = AllocateMatrix(KERNEL_SIZE, KERNEL_SIZE);
    N = AllocateMatrix(width, height);        
    P = AllocateMatrix(width, height);
    PS = AllocateMatrix(width, height);

    GenerateRandomMatrix(M);
    GenerateRandomMatrix(N);
    
    printf("Test for matrix size %dX%d = %d\n", height, width, height * width);

    // M * N pe device
    ConvolutionOnDevice(M, N, P);
    
    // M * N pe device cu memorie partajată
    ConvolutionOnDeviceShared(M, N, PS);

    
    //pentru măsurarea timpului de execuție pe CPU
    StopWatchInterface *kernelTime = NULL;
    sdkCreateTimer(&kernelTime);
    sdkResetTimer(&kernelTime);
    
    // calculează rezultatul pe CPU pentru comparație
    Matrix reference = AllocateMatrix(P.width, P.height);
    
    sdkStartTimer(&kernelTime);
    computeGold(reference.elements, M.elements, N.elements, N.height, N.width);
    
    sdkStopTimer(&kernelTime);
    printf ("Timp execuție CPU: %f ms\n", sdkGetTimerValue(&kernelTime));
     
        
    // verifică dacă rezultatul obținut pe device este cel așteptat
    int res = CompareMatrices(reference, P);
    printf("Test global %s\n", (1 == res) ? "PASSED" : "FAILED");
    fprintf(out, "Test global %s %s\n", argv[1], (1 == res) ? "PASSED" : "FAILED");
    
  
     
     

    // verifică dacă rezultatul obținut pe device cu memorie partajată este cel așteptat
    //  int ress = CompareMatrices(reference, PS);
    int ress = CompareMatrices(reference, PS);
    printf("Test shared %s\n", (1 == ress) ? "PASSED" : "FAILED");
    fprintf(out, "Test shared %s %s\n", argv[1], (1 == ress) ? "PASSED" : "FAILED");
    
    printf("\n");
   
    // Free matrices
    FreeMatrix(&M);
    FreeMatrix(&N);
    FreeMatrix(&P);
    FreeMatrix(&PS);

    fclose(f);
    fclose(out);
    return 0;
}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void ConvolutionOnDevice(const Matrix M, const Matrix N, Matrix P)
{
    Matrix Md, Nd, Pd; //matricele corespunzătoare de pe device
    size_t size;

    //pentru măsurarea timpului de execuție în kernel
    StopWatchInterface *kernelTime = NULL;
    sdkCreateTimer(&kernelTime);
    sdkResetTimer(&kernelTime);

    Md = AllocateDeviceMatrix(M.width, M.height);
    Nd = AllocateDeviceMatrix(N.width, N.height);
    Pd = AllocateDeviceMatrix(P.width, P.height);

    
    //matrice kernel
    size = M.width * M.height * sizeof(float); 
    hipMemcpy(Md.elements, M.elements, size, hipMemcpyHostToDevice);
    
    //matrice inițială de pe host
    size = N.width * N.height * sizeof(float); 
    hipMemcpy( Nd.elements, N.elements, size, hipMemcpyHostToDevice);

    // dimGrid: daca nu se imparte perfect o dimensiune, facem ceil() pe rezultat
    // ca sa acoperim toate cazurile
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((N.width + dimBlock.x - 1) / dimBlock.x, (N.height + dimBlock.y - 1) / dimBlock.y);
    

    sdkStartTimer(&kernelTime);

    ConvolutionKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd);

    hipDeviceSynchronize();
    sdkStopTimer(&kernelTime);
    printf ("Timp execuție kernel: %f ms\n", sdkGetTimerValue(&kernelTime));
    
    size = P.width * P.height * sizeof(float); 
    hipMemcpy(P.elements, Pd.elements, size, hipMemcpyDeviceToHost);
    
    FreeDeviceMatrix(&Md);
    FreeDeviceMatrix(&Nd);
    FreeDeviceMatrix(&Pd);
}


void ConvolutionOnDeviceShared(const Matrix M, const Matrix N, Matrix P)
{
    Matrix Md, Nd, Pd; //matricele corespunzătoare de pe device
    size_t size;

    //pentru măsurarea timpului de execuție în kernel
    StopWatchInterface *kernelTime = NULL;
    sdkCreateTimer(&kernelTime);
    sdkResetTimer(&kernelTime);
    
    Md = AllocateDeviceMatrix(M.width, M.height);
    Nd = AllocateDeviceMatrix(N.width, N.height);
    Pd = AllocateDeviceMatrix(P.width, P.height);

    //matrice kernel
    size = M.width * M.height * sizeof(float); 
    hipMemcpy(Md.elements, M.elements, size, hipMemcpyHostToDevice);
    
    //matrice inițială de pe host
    size = N.width * N.height * sizeof(float); 
    hipMemcpy( Nd.elements, N.elements, size, hipMemcpyHostToDevice);
    
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((N.width + dimBlock.x - 1) / dimBlock.x, (N.height + dimBlock.y - 1) / dimBlock.y);
    
    sdkStartTimer(&kernelTime);
    
    ConvolutionKernelShared<<<dimGrid, dimBlock>>>(Md, Nd, Pd);
        
    hipDeviceSynchronize();
    sdkStopTimer(&kernelTime);
    printf ("Timp execuție kernel cu memorie partajată: %f ms\n", sdkGetTimerValue(&kernelTime));
    
    size = P.width * P.height * sizeof(float); 
    hipMemcpy(P.elements, Pd.elements, size, hipMemcpyDeviceToHost);
    
    FreeDeviceMatrix(&Md);
    FreeDeviceMatrix(&Nd);
    FreeDeviceMatrix(&Pd);
}


// Alocă o matrice de dimensiune height*width pe device
Matrix AllocateDeviceMatrix(int width, int height)
{
    Matrix m;

    m.width = width;
    m.height = height;
    m.pitch = width;

    size_t size =  m.width * m.height * sizeof(float);
    hipMalloc( (void**) &(m.elements), size);

    return m;
}

// Alocă matrice pe host de dimensiune height*width
Matrix AllocateMatrix(int width, int height)
{
    Matrix M;
    M.width = M.pitch = width;
    M.height = height;
    int size = M.width * M.height;    
    M.elements = (float*) malloc(size*sizeof(float));
    return M;
}    

// Eliberează o matrice de pe device
void FreeDeviceMatrix(Matrix* M)
{
    hipFree(M->elements);
    M->elements = NULL;
}

// Eliberează o matrice de pe host
void FreeMatrix(Matrix* M)
{
    free(M->elements);
    M->elements = NULL;
}
